#include<hip/hip_runtime.h>
#include<cstdlib>
#include<cstdio>

#ifndef KERNELS
#define KERNELS

#define OP_NON 0
#define OP_ADD 1
#define OP_SUB 2
#define OP_MUL 3
#define OP_DIV 4

#define FN_SIGM 1 //sigmoid
#define FN_RELU 2 //relu
#define FN_DSIGM 3 //diffrentiation of sigmoid
#define FN_DRELU 4 //diffrentiation of relu

__device__ double operation(int op,double elem1,double elem2) {

	if (op == OP_ADD) return elem1 + elem2;
	else if(op == OP_SUB) return elem1 - elem2;
	else if(op == OP_MUL) return elem1 * elem2;
	else if(op == OP_DIV) return elem1 / (elem2 + 0.00000000001);
	else return elem1;
}

__device__ double function(int fn,double elem) {

	if (fn == FN_SIGM) return 1/(1 + exp(-1*elem));
	else if(fn == FN_RELU) return (elem > 0.00001 ? elem: 0);
	else if(fn == FN_DSIGM) {
		double sig = 1/(1 + exp(-1*elem));
		return sig*(1 - sig);
	}
	else if(fn == FN_DRELU) return(elem > 0.00001 ? 1 : 0);
	else return elem;

}

__global__ void gaxpy_kernel(int k_dim,double *mat1,double *mat2,double *res,double *matc = NULL, int c_row = 0, int c_col = 0) {

	int ri = blockIdx.x,rj = threadIdx.x;
	//printf("%d %d\n",blockDim.x,gridDim.x);
	double *p1 = mat1 + ri*k_dim, *p2 = mat2 + rj;
	double sum = 0;
	for(int k = 0; k < k_dim; k++) {
		//printf("%d %d %d %d %lf %lf \n",ri,k,k,rj,*p1,*p2);
		sum += (*p1) * (*p2);
		p1++;
		p2 += blockDim.x;
	}
	//printf("%d %d %lf\n",ri,rj,sum);
	if(matc == NULL) 
		res[ri * blockDim.x + rj] = sum;
	else {
		if(c_row == 1) res[ri * blockDim.x + rj] = sum + matc[rj];
		else if(c_col == 1) res[ri * blockDim.x + rj] = sum + matc[ri];
		else res[ri * blockDim.x + rj] = sum + matc[ri*blockDim.x + rj];
	}
}

__global__ void transpose_kernel(double *mat,double *tr_mat) {

	tr_mat[threadIdx.x*gridDim.x + blockIdx.x] = mat[blockIdx.x*blockDim.x + threadIdx.x];
}

__global__ void hadamard_kernel(double *mat1,double *mat2,double *hmat) {

	hmat[blockIdx.x*blockDim.x + threadIdx.x] = 
		mat1[blockIdx.x*blockDim.x + threadIdx.x] * mat2[blockIdx.x*blockDim.x + threadIdx.x];
		
}

__global__ void saxpy_kernel(double *mat1,double *mat2,double *res,double a = 1) {

	res[blockIdx.x*blockDim.x + threadIdx.x] = 
		a*mat1[blockIdx.x*blockDim.x + threadIdx.x] + mat2[blockIdx.x*blockDim.x + threadIdx.x];
}

__global__ void operate_kernel(double *mat1,double *res,double a,int op) {

	res[blockIdx.x*blockDim.x + threadIdx.x] = 
		operation(op,mat1[blockIdx.x*blockDim.x + threadIdx.x],a);
}

__global__ void function_kernel(double *mat1,double *res,int fn) {

	res[blockIdx.x*blockDim.x + threadIdx.x] = function(fn,mat1[blockIdx.x*blockDim.x + threadIdx.x]);
}

__global__ void reduction_kernel(double *mat1,double *res,int op,int dim,int axis) {

	double sum = 0;
	if(axis == 1) {
		for(int i = 0; i < dim; i++) 
			sum = operation(op,sum,mat1[i*blockDim.x + threadIdx.x]);
		res[threadIdx.x] = sum;
	}
	else {
		for(int i = 0; i < dim; i++)
			sum = operation(op,sum,mat1[threadIdx.x*dim + i]);
		if(axis == 3) {
			extern __shared__ double s[];
			__syncthreads();
			s[threadIdx.x] = sum;
			__syncthreads();
			if(threadIdx.x == 0) {
				sum = 0;
				for(int i = 0; i < blockDim.x; i++)
					sum = operation(op,sum,s[i]);
				res[0] = sum;
				//printf("%lf",sum);
			}
			__syncthreads();
			return;
		}
		res[threadIdx.x] = sum;
	}

}


#endif