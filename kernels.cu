#include<hip/hip_runtime.h>
#include<cstdlib>
#include<cstdio>

#ifndef KERNELS
#define KERNELS

__global__ void matmul_kernel(int k_dim,double *mat1,double *mat2,double *res) {

	int ri = blockIdx.x,rj = threadIdx.x;
	//printf("%d %d\n",blockDim.x,gridDim.x);
	double *p1 = mat1 + ri*k_dim, *p2 = mat2 + rj;
	double sum = 0;
	for(int k = 0; k < k_dim; k++) {
		//printf("%d %d %d %d %lf %lf \n",ri,k,k,rj,*p1,*p2);
		sum += (*p1) * (*p2);
		p1++;
		p2 += blockDim.x;
	}
	//printf("%d %d %lf\n",ri,rj,sum);
	res[ri * blockDim.x + rj] = sum;
}

__global__ void transpose_kernel(double *mat,double *tr_mat) {

	tr_mat[threadIdx.x*gridDim.x + blockIdx.x] = mat[blockIdx.x*blockDim.x + threadIdx.x];
	//printf("%lf ",tr_mat[threadIdx.x*gridDim.x + blockIdx.x]);
}

__global__ void hadamard_kernel(double *mat1,double *mat2,double *hmat) {

	hmat[blockIdx.x*blockDim.x + threadIdx.x] = 
		mat1[blockIdx.x*blockDim.x + threadIdx.x] * mat2[blockIdx.x*blockDim.x + threadIdx.x];
		
}
#endif