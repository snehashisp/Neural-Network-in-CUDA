#include "hip/hip_runtime.h"
#include"functions.cu"
#include<vector>

#ifndef NNET
#define NNET

/*
input is of dimension N x D : N = batch size and D = data dimension
*/

class neural_network {


	std :: vector<matrix *> weight_arr;
	std :: vector<matrix *> biases_arr;
	std :: vector<matrix *> output_arr;
	std :: vector<matrix *> activation_arr;
	std :: vector<matrix *> gradient_arr;

	matrix *mseLossDiff = NULL;
	matrix *mseLoss = NULL;
	matrix *singleLoss = NULL ;
	matrix *transPoseSpace = NULL;
	matrix *reductionMat = NULL;

	int batch_size, input_size, output_dim;
	int activation = FN_SIGM;
	int dactivation = FN_DSIGM;

	bool use_bias = false;

	public:

	void forward(matrix *data,bool updates = false) {

		data -> storeCuda();

		if(data -> height != batch_size || data -> width != input_size) {
			printf(" Input Data dimension mismatch required %d %d given %d %d",
				batch_size,input_size,data -> height,data -> width);
			return;
		}
		int i = 0;

		if(use_bias) 
			cuda_matmul(data,weight_arr[i],biases_arr[i],output_arr[i],updates);
		else
			cuda_matmul(data,weight_arr[i],output_arr[i],updates);
		cuda_function(output_arr[i],activation_arr[i],activation,updates);
		
		for(i += 1; i < weight_arr.size(); i++) {
			if(use_bias) 
				cuda_matmul(activation_arr[i-1],weight_arr[i],biases_arr[i],output_arr[i],updates);
			else
				cuda_matmul(activation_arr[i-1],weight_arr[i],output_arr[i],updates);
			cuda_function(output_arr[i],activation_arr[i],activation,updates);
		}	
	}

	void MSELossDiff(matrix *labels,bool updates = false) {

		labels -> storeCuda();

		matrix *final = activation_arr[activation_arr.size()-1];
		if(final -> height != labels -> height || final -> width != labels -> width) {
			printf(" Input Data dimension mismatch required %d %d given %d %d",
				final -> height,final -> width,labels -> height,labels -> width);
			return;
		}
		cuda_vecDiff(labels,final,mseLossDiff,updates);
	}

	double returnSingleLoss() {

		cuda_hadamard(mseLossDiff,mseLossDiff,mseLoss,true);
		cuda_reduce(mseLoss,singleLoss,OP_ADD,3,true);
		return singleLoss -> mat[0]/(batch_size * output_dim);
	}

	void backprop(matrix *data,double lrate,bool updates = false) {

		data -> storeCuda();

		int i = activation_arr.size() - 1;
		matrix *inp,*err = mseLossDiff;

		for(int i = activation_arr.size()-1; i >= 0; i--) {

			if(i == 0) inp = data;
			else inp = activation_arr[i-1];
			//error 
			cuda_function(output_arr[i],output_arr[i],dactivation);
			cuda_hadamard(err,output_arr[i],output_arr[i]);

			//weight update
			cuda_transpose(inp,transPoseSpace);
			transPoseSpace -> height = inp -> width;
			transPoseSpace -> width = inp -> height;
			cuda_matmul(transPoseSpace,output_arr[i],gradient_arr[i]);
			cuda_operation(gradient_arr[i],gradient_arr[i],lrate / batch_size,OP_MUL);


			//update bias
			if(use_bias) {
				cuda_reduce(err,reductionMat,OP_ADD,1);
				cuda_operation(reductionMat,reductionMat,lrate / batch_size,OP_MUL);
				cuda_vecADD(biases_arr[i],reductionMat,biases_arr[i]);
			}

			//partial error at prev level
			if(i > 0) {
				cuda_transpose(weight_arr[i],transPoseSpace);
				transPoseSpace -> height = weight_arr[i] -> width;
				transPoseSpace -> width = weight_arr[i] -> height;
				cuda_matmul(err,transPoseSpace,activation_arr[i-1]);
				err = activation_arr[i-1];
			}
		}

		for(int i = 0; i < gradient_arr.size(); i++) {
			cuda_vecADD(weight_arr[i],gradient_arr[i],weight_arr[i]);
		}

	}

	//public:

	void init(std::vector<int> nodeList,int bsize, 
		bool use_bias=true,double mean = 0,double std = 1) {

		//Re initialize all data structures if three exists a previous initialization
		for(int i = 0; i < weight_arr.size(); i++) {

			weight_arr[i] -> freeCuda();
			weight_arr[i] -> ~matrix();
			biases_arr[i] -> freeCuda();
			biases_arr[i] -> ~matrix();
			output_arr[i] -> freeCuda();
			output_arr[i] -> ~matrix();
			activation_arr[i] -> freeCuda();
			activation_arr[i] -> ~matrix();
		}

		weight_arr.clear();
		output_arr.clear();
		activation_arr.clear();
		biases_arr.clear();
		if(mseLoss) {
			mseLoss -> freeCuda();
			mseLoss -> ~matrix();
			mseLossDiff -> freeCuda();
			mseLossDiff -> ~matrix();
			singleLoss -> freeCuda();
			singleLoss -> ~matrix();
			transPoseSpace -> freeCuda();
			transPoseSpace -> ~matrix();
		}

		//new initialization 
		int max = 0;
		for(int i = 0; i < nodeList.size()-1; i++) {

			matrix *mat = new matrix, *omat = new matrix, 
				*amat = new matrix, *gmat = new matrix;
			mat -> init(nodeList[i],nodeList[i+1]);
			gaussianInitializer(mat,mean,std);

			if(use_bias) {
				matrix *bmat = new matrix;
				bmat -> init(1,nodeList[i+1]);
				gaussianInitializer(bmat,mean,std/2);
				biases_arr.push_back(bmat);
			}

			weight_arr.push_back(mat);
			output_arr.push_back(omat);
			activation_arr.push_back(amat);
			gradient_arr.push_back(gmat);

			if(nodeList[i] > max) max = nodeList[i];
		}

		mseLossDiff = new matrix;
		mseLoss = new matrix;
		singleLoss = new matrix;
		transPoseSpace = new matrix;
		reductionMat = new matrix;
		transPoseSpace -> init(max,bsize);

		batch_size = bsize;
		this -> activation = activation;
		this -> use_bias = use_bias;
		input_size = nodeList[0];
		output_dim = nodeList[nodeList.size()-1];

	}


	void trainModel(matrix *data, matrix *label,int epochs,double lrate) {
		data -> storeCuda();
		label -> storeCuda();
	}


	void print_weights() {

		for(int i = 0; i < weight_arr.size(); i++) {
			weight_arr[i] -> print_shape();
			weight_arr[i] -> print();
		}
	}

	void print_biases() {
		for(int i = 0; i < biases_arr.size(); i++) {
			biases_arr[i] -> print_shape();
			biases_arr[i] -> print();
		}
	}

	void print_outputs() {
		for(int i = 0; i < output_arr.size(); i++) {
			output_arr[i] -> print_shape();
			output_arr[i] -> print();
		}
	}

	void print_activations() {
		for(int i = 0; i < activation_arr.size(); i++) {
			activation_arr[i] -> print_shape();
			activation_arr[i] -> print();
		}
	}

	void printLossMat() {
		mseLoss -> print_shape();
		mseLoss -> print();
	}
};

#endif