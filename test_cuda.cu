#include<hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<cstdlib>
#include"functions.cu"




int main(int argc,char *argv[]) {

	matrix *mat1 = loadFromFile(argv[1]), *mat2 = loadFromFile(argv[2]);
	//mat1 -> print();
	mat2 -> print();
	//matrix *matm = matrix_multi(mat1,mat2);
	//matm -> print();
	matrix *mat3 = new matrix;
	//cuda_matmul(mat1,mat2,mat3,true);
	cuda_transpose(mat2,mat3,true);
	mat3 -> print();
	hipDeviceSynchronize();
	//while(1);
}