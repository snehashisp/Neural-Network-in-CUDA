#include<hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<cstdlib>
#include"neuralnet.cu"


using namespace std;

int main(int argc,char *argv[]) {

	matrix *mat1 = loadFromFile(argv[1]), *mat2 = loadFromFile(argv[2]);
	//matrix *mat3 = loadFromFile(argv[3]);
	mat1 -> print();
	cout << endl;
	mat2 -> print();
	cout << endl;
	// mat3 -> print();
	// cout << endl;
	//matrix *matm = matrix_multi(mat1,mat2);
	//matm -> print();
	matrix *mat4 = new matrix;
	// cuda_matmul(mat1,mat2,mat4,true);
	// mat4 -> print();
	// cout << endl;
	// cuda_matmul(mat1,mat2,mat3,mat4,true);
	// mat4 -> print();

	//cuda_function(mat2,mat4,FN_DSIGM,true);
	mat4 -> init(5,4);
	gaussianInitializer(mat4,0,1);
	mat4 -> print();
	//cuda_function(mat2,mat4,FN_DSIGM,true);
	//mat4 -> print();
	hipDeviceSynchronize();
	//while(1);
}	

