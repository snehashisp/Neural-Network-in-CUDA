#include<hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<cstdlib>
#include"neuralnet.cu"



using namespace std;

// int main(int argc,char *argv[]) {

// 	matrix *mat1 = loadFromFile(argv[1]), *mat2 = loadFromFile(argv[2]);
// 	//matrix *mat3 = loadFromFile(argv[3]);
// 	mat1 -> print();
// 	cout << endl;
// 	mat2 -> print();
// 	cout << endl;

// 	// cout << endl;
// 	//matrix *matm = matrix_multi(mat1,mat2);
// 	//matm -> print();
// 	matrix *mat4 = new matrix;
// 	mat4 -> init(6,6);
// 	cuda_transpose(mat1,mat4,true);
// 	mat4 -> height = mat1 -> width;
// 	mat4 -> width = mat1 -> height;
// 	mat4 -> print();
// 	cuda_transpose(mat2,mat4,true);
// 	mat4 -> height = mat2 -> width;
// 	mat4 -> width = mat2 -> height;
// 	mat4 -> print();
// 	matrix *mat5 = new matrix;
// 	cuda_matmul(mat2,mat4,mat5,true);
// 	mat5 -> print();
// 	// cout << endl;
// 	// cuda_matmul(mat1,mat2,mat3,mat4,true);
// 	// mat4 -> print();

// 	// //cuda_function(mat2,mat4,FN_DSIGM,true);
// 	// mat4 -> init(5,4);
// 	// gaussianInitializer(mat4,0,1);
// 	// mat4 -> print();
// 	// //cuda_function(mat2,mat4,FN_DSIGM,true);
// 	// //mat4 -> print();
// 	// hipDeviceSynchronize();
// 	// //while(1);
// }	

// int main() {

// 	neural_network nn;
// 	std :: vector<int> weights = {2,1,2};
// 	nn.init(weights,2);
// 	nn.print_weights();
// 	nn.print_biases();
// 	nn.print_outputs();
// 	nn.print_activations();

// 	matrix *inp = new matrix;
// 	inp->init(2,2);
// 	gaussianInitializer(inp,0,1);
// 	inp -> print();
// 	nn.forward(inp,true);
// 	nn.print_outputs();
// 	nn.print_activations();
// 	nn.MSELossDiff(inp,true);
// 	printf("Loss %lf\n",nn.returnSingleLoss());
// 	nn.printLossMat();
// 	printf("\n");

// 	gaussianInitializer(inp,0,1);
// 	inp -> print();
// 	nn.forward(inp,true);
// 	nn.print_outputs();
// 	nn.print_activations();
// 	nn.MSELossDiff(inp,true);
// 	printf("Loss %lf",nn.returnSingleLoss());
// 	nn.printLossMat();
// }

int main() {

	matrix *mat1 = new matrix, *mat2 = new matrix;
	readCSV(mat1,mat2,60000,785,false);

	matrix *mat3 = mat1 -> rowSlice(9,10);
	mat2 -> print_shape();
	mat3 -> print();
}