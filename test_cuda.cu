#include<hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<cstdlib>
#include"neuralnet.cu"



using namespace std;

// int main(int argc,char *argv[]) {

// 	// matrix *mat1 = loadFromFile(argv[1]), *mat2 = loadFromFile(argv[2]);
// 	// //matrix *mat3 = loadFromFile(argv[3]);
// 	// mat1 -> print();
// 	// cout << endl;
// 	// mat2 -> print();
// 	// cout << endl;

// 	// cout << endl;
// 	//matrix *matm = matrix_multi(mat1,mat2);
// 	//matm -> print();
// 	matrix *mat4 = new matrix;
// 	mat4 -> init(100,784);
// 	gaussianInitializer(mat4,0,1);
// 	mat4 -> print();
// 	// cuda_transpose(mat1,mat4,true);
// 	// mat4 -> height = mat1 -> width;
// 	// mat4 -> width = mat1 -> height;
// 	// mat4 -> print();
// 	// cuda_transpose(mat2,mat4,true);
// 	// mat4 -> height = mat2 -> width;
// 	// mat4 -> width = mat2 -> height;
// 	// mat4 -> print();
// 	matrix *mat5 = new matrix;
// 	cuda_reduce(mat4,mat5,OP_ADD,1,true);
// 	printf("\nF");
// 	mat5 -> print();
// 	// cout << endl;
// 	// cuda_matmul(mat1,mat2,mat3,mat4,true);
// 	// mat4 -> print();

// 	// //cuda_function(mat2,mat4,FN_DSIGM,true);
// 	// mat4 -> init(5,4);
// 	// gaussianInitializer(mat4,0,1);
// 	// mat4 -> print();
// 	// //cuda_function(mat2,mat4,FN_DSIGM,true);
// 	// //mat4 -> print();
// 	// hipDeviceSynchronize();
// 	// //while(1);
// }	

//int main() {

// 	neural_network nn;
// 	std :: vector<int> weights = {784,10,784};
// 	nn.init(weights,100);
// 	//nn.print_weights();
// 	//nn.print_biases();
// 	//nn.print_outputs();
// 	//nn.print_activations();

// 	matrix *inp = new matrix;
// 	inp->init(100,784);
// 	gaussianInitializer(inp,5,1);
// 	//inp -> print();
// 	int k = 10;
// 	while(k--) {
// 		nn.forward(inp);
// 		//nn.print_outputs();
// 		//nn.print_activations();
// 		nn.MSELossDiff(inp);
// 		printf("Loss %lf\n",nn.returnSingleLoss());
// 		nn.backprop(inp,0.01);
// 	}
// 	//nn.print_weights();
// 	//nn.print_biases();
// 	// //nn.printLossMat();
// 	// printf("\n");

// }

int main() {

	matrix *mat1 = new matrix, *mat2 = new matrix;
	readCSV(mat1,mat2,60000,785,false);
	cuda_operation(mat1,mat1,255,OP_DIV,true);
	neural_network nn;
	std :: vector<int> weights = {784,50,784};
	nn.init(weights,1000,false);
	nn.trainModel(mat1,mat1,100,0.01,2);
	matrix *emat = nn.encode(mat1);
	storeAsCSV(emat,"results.csv");
	// matrix *mat3 = new matrix, *mat4 = new matrix;
	// mat1 -> rowSlice(mat3,0,100);
	// neural_network nn;
	// std :: vector<int> weights = {784,200,784};
	// nn.init(weights,1000,false,0,1);
	// int k = 10;
	// while(k--) {
	// 	nn.forward(mat3);
	// 	//nn.print_activations();
	// 	nn.MSELossDiff(mat3,true);
	// 	printf("Loss %6.10lf\n",nn.returnSingleLoss());
	// 	//nn.printLossMat();
	// 	//break;
	// 	nn.backprop(mat3,0.1);
	// }
	// hipDeviceSynchronize();
}