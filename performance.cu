#include"neuralnet.cu"
#include<chrono>

using namespace std;
double *timeMulti(matrix *mat1,matrix *mat2) {

	double *timearr = new double[2];
	auto t_start = std::chrono::high_resolution_clock::now();
	matrix *tmat = matrix_multi(mat1,mat2);
	auto t_end = std::chrono::high_resolution_clock::now();
	timearr[0] = std::chrono::duration<double, std::milli>(t_end-t_start).count();

	t_start = std::chrono::high_resolution_clock::now();
	matrix *cmat = new matrix();
	cuda_matmul(mat1,mat2,cmat);
	t_end = std::chrono::high_resolution_clock::now();
	cmat -> updateCuda();
	timearr[1] = std::chrono::duration<double, std::milli>(t_end-t_start).count();

	cmat -> freeCuda();
	cmat -> ~matrix();
	tmat -> ~matrix();
	return timearr;
}



double *timeHadamard(matrix *mat1,matrix *mat2) {

	double *timearr = new double[2];
	auto t_start = std::chrono::high_resolution_clock::now();
	matrix *tmat = point_multi(mat1,mat2);
	auto t_end = std::chrono::high_resolution_clock::now();
	timearr[0] = std::chrono::duration<double, std::milli>(t_end-t_start).count();

	t_start = std::chrono::high_resolution_clock::now();
	matrix *cmat = new matrix();
	cuda_hadamard(mat1,mat2,cmat);
	t_end = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();
	//cmat -> updateCuda();
	timearr[1] = std::chrono::duration<double, std::milli>(t_end-t_start).count();

	cmat -> freeCuda();
	cmat -> ~matrix();
	tmat -> ~matrix();
	return timearr;
}

int main(int s,char *argv[]) {

	int a = atoi(argv[1]),b = atoi(argv[2]),c = atoi(argv[3]);
	matrix *results = new matrix;
	results -> init(b + 1,3);


	// for(int i = a; i <= a + b; i += c) {
	// 	matrix *mat1 = new matrix, *mat2 = new matrix;
	// 	mat1 -> init(i,i);
	// 	mat2 -> init(i,i);
	// 	gaussianInitializer(mat1);
	// 	gaussianInitializer(mat2);
	// 	double *res = timeMulti(mat1,mat2);
	// 	cout << i <<","<<res[0]<<","<<res[1]<<endl;
	// 	results -> mat[(i - a)*2] = i;
	// 	results -> mat[(i - a)*2 + 1] = res[0];
	// 	results -> mat[(i - a)*2 + 2] = res[1];
	// 	mat1 -> ~matrix();
	// 	mat2 -> ~matrix();
	// }


	for(int i = a; i <= a + b; i += c) {
		matrix *mat1 = new matrix, *mat2 = new matrix;
		mat1 -> init(i,i);
		mat2 -> init(i,i);
		gaussianInitializer(mat1);
		gaussianInitializer(mat2);
		double *res = timeHadamard(mat1,mat2);
		cout << i <<","<<res[0]<<","<<res[1]<<endl;
		results -> mat[(i - a)*2] = i;
		results -> mat[(i - a)*2 + 1] = res[0];
		results -> mat[(i - a)*2 + 2] = res[1];
		mat1 -> ~matrix();
		mat2 -> ~matrix();
	}

}
